#include "flexflow/utils/cuda_helper.h"
#include "llama.h"

void DataLoader::load_input(Task const *task,
                            std::vector<PhysicalRegion> const &regions,
                            Context ctx,
                            Runtime *runtime) {
  fprintf(stderr, "----------start load input--------------");                                        
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  SampleIdxs *meta = (SampleIdxs *)task->local_args;

  TensorAccessorR<float, 4> full_input(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<float, 4> batch_input(regions[1],
                                            task->regions[1],
                                            FID_DATA,
                                            ctx,
                                            runtime,
                                            false /*readOutput*/);
//   int const *full_input_ptr = helperGetTensorPointerRO<int>(
//       regions[0], task->regions[0], FID_DATA, ctx, runtime);
//   int *batch_input_ptr = helperGetTensorPointerWO<int>(
//       regions[1], task->regions[1], FID_DATA, ctx, runtime);

  Domain full_input_domain = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  Domain batch_input_domain = runtime->get_index_space_domain(
      ctx, task->regions[1].region.get_index_space());

  coord_t token_dim =
      batch_input_domain.hi()[0] - batch_input_domain.lo()[0] + 1;
  coord_t sequence_length =
      batch_input_domain.hi()[1] - batch_input_domain.lo()[1] + 1;
  coord_t batch_size =
      batch_input_domain.hi()[2] - batch_input_domain.lo()[2] + 1;

  // FIXME: currently assume continous indices
  assert(meta->num_samples <= batch_size);
  for (int i = 1; i < meta->num_samples; i++) {
    assert(meta->idxs[i] == meta->idxs[0] + i);
  }
  // pad inputs if needed (this is really only useful for debugging)
  if (meta->num_samples < batch_size) {
    checkCUDA(hipMemset(batch_input.ptr +
                             token_dim * sequence_length * meta->num_samples,
                         0,
                         token_dim * sequence_length *
                             (batch_size - meta->num_samples) * sizeof(float)));
  }
  coord_t start_idx = meta->idxs[0];
  assert(batch_input_domain.get_volume() % token_dim * sequence_length *
             batch_size ==
         0);
  assert(batch_input_domain.get_volume() % batch_size == 0);
  size_t size_to_copy =
      (batch_input_domain.get_volume() / batch_size) * meta->num_samples;
  float const *input_zc =
      full_input.ptr + start_idx * token_dim * sequence_length;
  copy_kernel<<<GET_BLOCKS(size_to_copy), CUDA_NUM_THREADS>>>(
      batch_input.ptr, input_zc, size_to_copy);
  checkCUDA(hipDeviceSynchronize());
}

void DataLoader::load_label(Task const *task,
                            std::vector<PhysicalRegion> const &regions,
                            Context ctx,
                            Runtime *runtime) {
   fprintf(stderr, "----------start load label--------------");                                       
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  SampleIdxs *meta = (SampleIdxs *)task->local_args;
  TensorAccessorR<int, 4> full_label(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<int, 4> batch_label(regions[1],
                                            task->regions[1],
                                            FID_DATA,
                                            ctx,
                                            runtime,
                                            false /*readOutput*/);
  int const *full_label_ptr = helperGetTensorPointerRO<int>(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  int *batch_label_ptr = helperGetTensorPointerWO<int>(
      regions[1], task->regions[1], FID_DATA, ctx, runtime);
  Domain full_label_domain = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  Domain batch_label_domain = runtime->get_index_space_domain(
      ctx, task->regions[1].region.get_index_space());
  coord_t label_dim =
      batch_label_domain.hi()[0] - batch_label_domain.lo()[0] + 1;
  coord_t sequence_length =
      batch_label_domain.hi()[1] - batch_label_domain.lo()[1] + 1;
  coord_t batch_size =
      batch_label_domain.hi()[2] - batch_label_domain.lo()[2] + 1;
  // FIXME: currently assume continous indices
  assert(meta->num_samples <= batch_size);
  for (int i = 1; i < meta->num_samples; i++) {
    assert(meta->idxs[i] == meta->idxs[0] + i);
  }
  if (meta->num_samples < batch_size) {
    checkCUDA(hipMemset(batch_label.ptr +
                             label_dim * sequence_length * meta->num_samples,
                         0,
                         label_dim * sequence_length *
                             (batch_size - meta->num_samples) * sizeof(int)));
  }
  assert(batch_label_domain.get_volume() % label_dim * sequence_length *
             batch_size ==
         0);
  assert(batch_label_domain.get_volume() % batch_size == 0);
  coord_t start_idx = meta->idxs[0];
  size_t size_to_copy =
      (batch_label_domain.get_volume() / batch_size) * meta->num_samples;
  int const *input_zc =
      full_label.ptr + start_idx * label_dim * sequence_length;
  copy_kernel<<<GET_BLOCKS(size_to_copy), CUDA_NUM_THREADS>>>(
      batch_label.ptr, input_zc, size_to_copy);
  checkCUDA(hipDeviceSynchronize());
}

void DataLoader::load_pos(Task const *task,
                            std::vector<PhysicalRegion> const &regions,
                            Context ctx,
                            Runtime *runtime) {
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  SampleIdxs *meta = (SampleIdxs *)task->local_args;
  TensorAccessorR<float, 4> full_pos(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<float, 4> batch_pos(regions[1],
                                            task->regions[1],
                                            FID_DATA,
                                            ctx,
                                            runtime,
                                            false /*readOutput*/);
  
   Domain full_pos_domain = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
   Domain batch_pos_domain = runtime->get_index_space_domain(
      ctx, task->regions[1].region.get_index_space());

  coord_t token_dim =
      batch_pos_domain.hi()[0] - batch_pos_domain.lo()[0] + 1;
  coord_t sequence_length =
      batch_pos_domain.hi()[1] - batch_pos_domain.lo()[1] + 1;
  coord_t batch_size =
      batch_pos_domain.hi()[2] - batch_pos_domain.lo()[2] + 1;
 
  assert(meta->num_samples <= batch_size);
  for (int i = 1; i < meta->num_samples; i++) {
    assert(meta->idxs[i] == meta->idxs[0] + i);
  }
  if (meta->num_samples < batch_size) {
    checkCUDA(hipMemset(batch_pos.ptr+
                             token_dim * sequence_length * meta->num_samples,
                         0,
                         token_dim * sequence_length *
                             (batch_size - meta->num_samples) * sizeof(float)));
  }
  coord_t start_idx = meta->idxs[0];
  assert(batch_pos_domain.get_volume() % token_dim * sequence_length *
             batch_size ==
         0);
  assert(batch_pos_domain.get_volume() % batch_size == 0);
  size_t size_to_copy =
      (batch_pos_domain.get_volume() / batch_size) * meta->num_samples;
  float const *input_zc =
      full_pos.ptr + start_idx * token_dim * sequence_length;
  copy_kernel<<<GET_BLOCKS(size_to_copy), CUDA_NUM_THREADS>>>(
      batch_pos.ptr, input_zc, size_to_copy);
  checkCUDA(hipDeviceSynchronize());
}